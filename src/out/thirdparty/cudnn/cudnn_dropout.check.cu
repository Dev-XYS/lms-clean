/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/**************** Snippet ****************/
void Snippet(int x0) {
  printf("setting input image\n");
  float* x1 = (float*)malloc(81 * sizeof(float));
  int x2 = 0;
  while (x2 != 81) {
    int x3 = x2;
    x1[x3] = (float)x3;
    x2 = x2 + 1;
  }
  printf("setting up device\n");
  CUDA_CALL(hipSetDevice(0));
  printf("create handle\n");
  hipdnnHandle_t x4;
  CUDNNCHECK(hipdnnCreate(&x4));
  printf("create input descriptor\n");
  hipdnnTensorDescriptor_t x5;
  CUDNNCHECK(hipdnnCreateTensorDescriptor(&x5));
  CUDNNCHECK(hipdnnSetTensor4dDescriptor(x5, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 9, 9));
  printf("ask CuDNN for memory of reserve space and states\n");
  size_t x6 = 0;
  CUDNNCHECK(cudnnDropoutGetReserveSpaceSize(x5, &x6));
  size_t x7 = 0;
  CUDNNCHECK(hipdnnDropoutGetStatesSize(x4, &x7));
  printf("reserve_bytes: %zu, states_bytes: %zu\n", x6, x7);
  printf("allocate memory for states and reserve space\n");
  float* x8 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x8, x7));
  float* x9 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x9, x6));
  printf("create dropout descriptor\n");
  hipdnnDropoutDescriptor_t x10;
  CUDNNCHECK(hipdnnCreateDropoutDescriptor(&x10));
  CUDNNCHECK(hipdnnSetDropoutDescriptor(x10, x4, 0.5, x8, x7, 1));
  printf("create output descriptor\n");
  hipdnnTensorDescriptor_t x11;
  CUDNNCHECK(hipdnnCreateTensorDescriptor(&x11));
  CUDNNCHECK(hipdnnSetTensor4dDescriptor(x11, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 9, 9));
  printf("allocate memory for input image and copy\n");
  float* x12 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x12, (size_t)(81 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x12, x1, (size_t)(81 * sizeof(float)), hipMemcpyHostToDevice));
  printf("allocate memory for output image\n");
  float* x13 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x13, (size_t)(81 * sizeof(float))));
  CUDA_CALL(hipMemset(x13, 0, (size_t)(81 * sizeof(float))));
  CUDNNCHECK(cudnnDropoutForward(x4, x10, x5, x12, x11, x13, x9, x6));
  float* x14 = (float*)malloc(81 * sizeof(float));
  CUDA_CALL(hipMemcpy(x14, x13, (size_t)(81 * sizeof(float)), hipMemcpyDeviceToHost));
  printf("print output:\n");
  int x15 = 0;
  while (x15 != 81) {
    printf("%f, ", x14[x15]);
    x15 = x15 + 1;
  }
  CUDA_CALL(hipFree(x12));
  CUDA_CALL(hipFree(x13));
  CUDA_CALL(hipFree(x9));
  CUDA_CALL(hipFree(x8));
  CUDNNCHECK(hipdnnDestroyTensorDescriptor(x5));
  CUDNNCHECK(hipdnnDestroyTensorDescriptor(x11));
  CUDNNCHECK(hipdnnDestroyDropoutDescriptor(x10));
  CUDNNCHECK(hipdnnDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
