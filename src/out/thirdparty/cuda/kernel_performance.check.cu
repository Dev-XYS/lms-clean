#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x9(int x10, float x11, float* x12, float* x13) {
  int x14 = blockIdx.x * blockDim.x * threadIdx.x;
  if (x14 < x10) x13[x14] = x11 * x12[x14] + x13[x14];
}
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(4096 * sizeof(float));
  float* x2 = (float*)malloc(4096 * sizeof(float));
  float* x3 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x3, (size_t)(4096 * sizeof(float))));
  float* x4 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x4, (size_t)(4096 * sizeof(float))));
  int x5 = 0;
  while (x5 != 4096) {
    int x6 = x5;
    x1[x6] = 1.0;
    x2[x6] = 2.0;
    x5 = x5 + 1;
  }
  hipEvent_t x7;
  hipEvent_t x8;
  CUDA_CALL(hipEventCreate(&x7));
  CUDA_CALL(hipEventCreate(&x8));
  CUDA_CALL(hipMemcpy(x3, x1, (size_t)(4096 * sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(x4, x2, (size_t)(4096 * sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipEventRecord(x7));
  x9<<<dim3(8, 1, 1), dim3(512, 1, 1)>>>(4096, 2.0, x3, x4);
  CUDA_CALL(hipEventRecord(x8));
  CUDA_CALL(hipMemcpy(x2, x4, (size_t)(4096 * sizeof(float)), hipMemcpyDeviceToHost));
  CUDA_CALL(hipEventSynchronize(x8));
  float x15 = 0.0;
  CUDA_CALL(hipEventElapsedTime(&x15, x7, x8));
  float x16 = 0.0;
  int x17 = 0;
  while (x17 != 4096) {
    float x18 = abs(x2[x17] - 4.0);
    if (x18 > x16) x16 = x18;
    x17 = x17 + 1;
  }
  printf("Max error: %f\n", x16);
  printf("Time: %f\n", x15);
  printf("Effective Bandwidth (GB/s): %f\n", (double)(49152.0 / x15) / 1000000.0);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
