#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/**************** Snippet ****************/
void Snippet(int x0) {
  // Sanity check only, not runnable code
  __shared__ int* x1[1][2];
  printf("%d", x1[0][1]);
  __shared__ int** x2[1][2][3];
  printf("%d", x2[0][1][0]);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
