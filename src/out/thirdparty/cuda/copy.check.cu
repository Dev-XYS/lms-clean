#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x7(int* x8, int* x9) {
  // Cuda Matrix Copy
  // arg0: 2D Input Matrix (n x n) where n is a multiple of 32
  // arg1: 2D Output Matrix (n x n) where n is a multiple of 32
  int x10 = blockIdx.x * 32 + threadIdx.x;
  int x11 = blockIdx.y * 32 + threadIdx.y;
  int x12 = gridDim.x * 32;
  int x13 = 0;
  while (x13 < 32) {
    int x14 = (x11 + x13) * x12 + x10;
    x9[x14] = x8[x14];
    x13 = x13 + 8;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(4096 * sizeof(int));
  int* x2 = (int*)malloc(4096 * sizeof(int));
  int x3 = 0;
  while (x3 != 4096) {
    int x4 = x3;
    x1[x4] = x4 + 1;
    x3 = x3 + 1;
  }
  int* x5 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x5, (size_t)(4096 * sizeof(int))));
  int* x6 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x6, (size_t)(4096 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x5, x1, (size_t)(4096 * sizeof(int)), hipMemcpyHostToDevice));
  x7<<<dim3(2, 2, 1), dim3(32, 8, 1)>>>(x5, x6);
  CUDA_CALL(hipMemcpy(x2, x6, (size_t)(4096 * sizeof(int)), hipMemcpyDeviceToHost));
  int x15 = 0;
  while (x15 != 64) {
    int x16 = 0;
    int x17 = 64 * x15;
    while (x16 != 64) {
      int x18 = x17 + x16;
      if (x1[x18] != x2[x18]) {
        printf("Copy Incorrect!\n");
        fflush(stdout); fflush(stderr); exit(1);
      }
      x16 = x16 + 1;
    }
    x15 = x15 + 1;
  }
  printf("Copy Correct\n");
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
