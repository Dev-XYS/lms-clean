#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "scanner_header.h"
/************* Functions **************/
__global__ void x7(float* x8, float* x9, int* x10, float x11, int x12, int x13, int x14, int x15, int x16) {
  // this is the cuda masked fill kernel.
  // `ijSwapped` is true if dim0 > dim1, in this case dim0 and dim1 are swapped
  // arg0: 2D in: input array of size `input_size`
  // arg1: 2D out: output array of size `input_size`
  // arg2: 2D out: output array of size input_size`
  // arg3: value: the value to fill
  // arg4: dim0_shape: shape of dim0
  // arg5: dim1_shape: shape of dim1
  // arg6: dim0_stride: stride of dim0
  // arg7: dim1_stride: stride of dim1
  // arg8: input_size: number of elements of arg0, arg1 and arg2
  int x17 = blockIdx.x * blockDim.x + threadIdx.x;
  int x18 = x17;
  int x19 = blockDim.x * gridDim.x;
  int x20 = x17 / x14;
  int x21 = x20;
  int x22 = x20 * x14;
  int x23 = x17 - x22;
  int x24 = x23 / x15;
  int x25 = x24;
  int x26 = x24 * x15;
  int x27 = x22 + x26 + (x23 - x26);
  while (x27 < x16) {
    x9[x27] = x10[x25 % x13 * x12 + x21 % x12] == 0 ? x8[x27] : x11;
    int x28 = x18 + x19;
    x18 = x28;
    int x29 = x28 / x14;
    x21 = x29;
    int x30 = x29 * x14;
    int x31 = x28 - x30;
    int x32 = x31 / x15;
    x25 = x32;
    int x33 = x32 * x15;
    x27 = x30 + x33 + (x31 - x33);
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(4096 * sizeof(float));
  scan_float("golden/emaskedFill/input.data", x1, 4096);
  float* x2 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x2, (size_t)(4096 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x2, x1, (size_t)(4096 * sizeof(float)), hipMemcpyHostToDevice));
  int* x3 = (int*)malloc(4096 * sizeof(int));
  scan_int("golden/emaskedFill/mask.data", x3, 4096);
  int* x4 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x4, (size_t)(4096 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x4, x3, (size_t)(4096 * sizeof(int)), hipMemcpyHostToDevice));
  float* x5 = (float*)malloc(4096 * sizeof(float));
  float* x6 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x6, (size_t)(4096 * sizeof(float))));
  x7<<<dim3(8, 1, 1), dim3(512, 1, 1)>>>(x2, x6, x4, 0.0, 64, 64, 64, 1, 4096);
  CUDA_CALL(hipMemcpy(x5, x6, (size_t)(4096 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array("golden/emaskedFill/output.data", x5, 4096);
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
