#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
/************* Functions **************/
__global__ void x7(int* x8, int* x9) {
  // Cuda Transpose Naive
  // arg0: 2D Input Matrix (n x n) where n is a multiple of 32
  // arg1: 2D Output Matrix (n x n) where n is a multiple of 32
  int x10 = blockIdx.x * 32 + threadIdx.x;
  int x11 = blockIdx.y * 32 + threadIdx.y;
  int x12 = gridDim.x * 32;
  int x13 = 0;
  int x14 = x10 * x12;
  while (x13 < 32) {
    int x15 = x11 + x13;
    x9[x14 + x15] = x8[x15 * x12 + x10];
    x13 = x13 + 8;
  }
}
/**************** Snippet ****************/
void Snippet(int x0) {
  int* x1 = (int*)malloc(4096 * sizeof(int));
  int* x2 = (int*)malloc(4096 * sizeof(int));
  int x3 = 0;
  while (x3 != 4096) {
    int x4 = x3;
    x1[x4] = x4 + 1;
    x3 = x3 + 1;
  }
  int* x5 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x5, (size_t)(4096 * sizeof(int))));
  int* x6 = (int*)malloc(0 * sizeof(int));
  CUDA_CALL(hipMalloc(&x6, (size_t)(4096 * sizeof(int))));
  CUDA_CALL(hipMemcpy(x5, x1, (size_t)(4096 * sizeof(int)), hipMemcpyHostToDevice));
  x7<<<dim3(2, 2, 1), dim3(32, 8, 1)>>>(x5, x6);
  CUDA_CALL(hipMemcpy(x2, x6, (size_t)(4096 * sizeof(int)), hipMemcpyDeviceToHost));
  int x16 = 0;
  while (x16 != 64) {
    int x17 = x16;
    int x18 = 0;
    int x19 = 64 * x17;
    while (x18 != 64) {
      int x20 = x18;
      if (x1[x19 + x20] != x2[64 * x20 + x17]) {
        printf("Transpose Incorrect!\n");
        fflush(stdout); fflush(stderr); exit(1);
      }
      x18 = x18 + 1;
    }
    x16 = x16 + 1;
  }
  printf("Transpose Correct\n");
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
