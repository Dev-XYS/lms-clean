#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x13(float* x14, float x15, int x16) {
  int x17 = gridDim.x * blockDim.x;
  int x18 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x18 < x16) {
    x14[x18] = x15;
    x18 = x18 + x17;
  }
}
__global__ void x21(float* x22, float* x23, int x24) {
  // begin generating kernel function for ACCUM of type Float
  int x25 = gridDim.x * blockDim.x;
  int x26 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x26 < x24) {
    int x27 = x26;
    x22[x27] = x22[x27] + x23[x27];
    x26 = x26 + x25;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x28(float* x29, float* x30, float* x31, int x32) {
  // begin generating kernel function for SGD of type Float
  int x33 = gridDim.x * blockDim.x;
  int x34 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x34 < x32) {
    int x35 = x34;
    float x36 = x31[x35] * 0.5 + x30[x35];
    x29[x35] = x29[x35] - x36 * 1.0E-4;
    x31[x35] = x36;
    x34 = x34 + x33;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  int x3 = MPI_Comm_size(MPI_COMM_WORLD, &x1);
  MPICHECK(x3);
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x4;
  NCCLCHECK(ncclGetUniqueId(&x4));
  MPICHECK(MPI_Bcast(&x4, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x5;
  NCCLCHECK(ncclCommInitRank(&x5, x1, x4, x2));
  hipStream_t x6;
  CUDA_CALL(hipStreamCreateWithFlags(&x6, hipStreamNonBlocking));
  int x7 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x8;
  CUDNNCHECK(hipdnnCreate(&x8));
  // end setting up the CUDNN environment
  // begin initializing random GPU array of size 9 and type Float at device (pre-rename) x39
  float* x9 = (float*)malloc(9 * sizeof(float));
  int x10 = 0;
  while (x10 != 9) {
    x9[x10] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x10 = x10 + 1;
  }
  CUDA_CALL(hipSetDevice(x7));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(9 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x11, x9, (size_t)(9 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 9 and type Float at device (pre-rename) x39
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x12 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x12, (size_t)(9 * sizeof(float))));
  x13<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x12, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x7));
  float* x19 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x19, (size_t)(9 * sizeof(float))));
  x13<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x19, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  int x20 = 0;
  while (x20 != 10) {
    // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x37 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x37, (size_t)(9 * sizeof(float))));
    x13<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x37, 0, 9);
    // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
    // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x7));
    float* x38 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x38, (size_t)(9 * sizeof(float))));
    x13<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x38, 1, 9);
    // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
    // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x142 and addition_operand x155
    CUDA_CALL(hipSetDevice(x7));
    x21<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x37, x38, 9);
    // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x142 and addition_operand x155
    // begin creating and setting tensor descriptor of shape List(2, 1, 3, 3)
    hipdnnTensorDescriptor_t x39;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x39));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x39, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 3, 3));
    // end creating and setting tensor descriptor
    // begin allocating gpu array for the gradient of input of softmax
    CUDA_CALL(hipSetDevice(x7));
    float* x40 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x40, (size_t)(18 * sizeof(float))));
    // end allocating gpu array for the gradient of input of softmax
    // begin softmax backward pass
    float x41 = 1.0;
    float x42 = 0.0;
    CUDNNCHECK(hipdnnSoftmaxBackward(x8, HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_INSTANCE, &x41, &x39, &x11, x39, &x37, &x42, x39, &x40));
    // end softmax backward pass
    // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x86 and addition_operand x214
    CUDA_CALL(hipSetDevice(x7));
    x21<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x12, x40, 9);
    // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x86 and addition_operand x214
    // begin computing SGD on GPU for size 9 and type Float at device (pre-name) x39 with weight x70, grad x86, and momentum x124
    CUDA_CALL(hipSetDevice(x7));
    x28<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x12, x19, 9);
    // end computing SGD on GPU for size 9 and type Float at device (pre-name) x39 with weight x70, grad x86, and momentum x124
    x20 = x20 + 1;
  }
  // Only declare recv buffer if this is the root
  bool x43 = x7 == 0;
  float* x44 = x43 ? ({
    float* x45 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x45, (size_t)(18 * sizeof(float))));
    x45;
  }) : ({
    float* x46 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x46, (size_t)0));
    x46;
  });
  // Gather by groups of NCCL send/recv
  NCCLCHECK(ncclGroupStart());
  ncclResult_t x47 = ncclSend(x11, (size_t)18, ncclFloat32, 0, x5, x6);
  NCCLCHECK(x47);
  if (x43) {
    int x48 = x1;
    int x49 = 0;
    while (x49 != x48) {
      int x50 = x49;
      NCCLCHECK(ncclRecv(x44 + x50 * 9, (size_t)18, ncclFloat32, x50, x5, x6));
      x49 = x49 + 1;
    }
  }
  NCCLCHECK(ncclGroupEnd());
  // print the array only if this is the root
  if (x43) {
    // begin copying GPU array x298 to CPU and print for size 18 and type Float
    float* x51 = (float*)malloc(18 * sizeof(float));
    CUDA_CALL(hipMemcpy(x51, x44, (size_t)(18 * sizeof(float)), hipMemcpyDeviceToHost));
    int x52 = 0;
    while (x52 != 18) {
      printf("%f ", x51[x52]);
      x52 = x52 + 1;
    }
    printf("\n");
    // end copying GPU array x298 to CPU and print for size 18 and type Float
  }
  printf("compile");
  CUDNNCHECK(hipdnnDestroy(x8));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x5));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
