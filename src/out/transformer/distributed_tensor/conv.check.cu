#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for SGD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    float x35 = x30[x34] * 0.5 + x29[x34];
    x28[x34] = x28[x34] - x35 * 1.0E-4;
    x30[x34] = x35;
    x33 = x33 + x32;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x7;
  CUDNNCHECK(hipdnnCreate(&x7));
  // end setting up the CUDNN environment
  // begin initializing random GPU array of size 9 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(9 * sizeof(float));
  int x9 = 0;
  while (x9 != 9) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(9 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(9 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 9 and type Float at device (pre-rename) x39
  NCCLCHECK(ncclAllReduce(x10, x10, (size_t)(9 * sizeof(float)), ncclFloat32, ncclSum, x4, x5));
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(9 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(9 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 9);
  // end initializing fixed GPU array of size 9 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing random GPU array of size 0 and type Float at device (pre-rename) x39
    float* x36 = (float*)malloc(0 * sizeof(float));
    int x37 = 0;
    while (x37 != 0) {
      x36[x37] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
      x37 = x37 + 1;
    }
    CUDA_CALL(hipSetDevice(x6));
    float* x38 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x38, (size_t)0));
    CUDA_CALL(hipMemcpy(x38, x36, (size_t)0, hipMemcpyHostToDevice));
    // end initializing random GPU array of size 0 and type Float at device (pre-rename) x39
    // begin initializing fixed GPU array of size 0 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x6));
    float* x39 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x39, (size_t)0));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x39, 1, 0);
    // end initializing fixed GPU array of size 0 and type Float and device (pre-rename) x39
    hipdnnTensorDescriptor_t x40;
    hipdnnCreateTensorDescriptor(&x40);
    hipdnnSetTensor4dDescriptor(x40, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 3, 3);
    hipdnnConvolutionDescriptor_t x41;
    hipdnnCreateConvolutionDescriptor(&x41);
    hipdnnSetConvolution2dDescriptor(x41, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);
    CUDA_CALL(hipSetDevice(x6));
    float* x42 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x42, (size_t)(9 * sizeof(float))));
    hipdnnConvolutionBwdFilterAlgoPerf_t x43;
    hipdnnFindConvolutionBackwardFilterAlgorithm(x7, x40, x40, x41, x40, 1, &0, &x43);
    CUDA_CALL(hipSetDevice(x6));
    float* x44 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x44, (size_t)0));
    float x45 = 1.0;
    float x46 = 0.0;
    hipdnnConvolutionBackwardData(x7, x45, &x40, x38, x40, x39, x41, x43.algo, x44, &(size_t)0, x46, x40, x42);
    // begin computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x205
    CUDA_CALL(hipSetDevice(x6));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x42, 9);
    // end computing ACCUM on GPU for size 9 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x205
    // begin computing SGD on GPU for size 9 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    CUDA_CALL(hipSetDevice(x6));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 9);
    // end computing SGD on GPU for size 9 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    x19 = x19 + 1;
  }
  if (x6 == 0) {
    // begin copying GPU array x70 to CPU and print for size 9 and type Float
    float* x47 = (float*)malloc(9 * sizeof(float));
    CUDA_CALL(hipMemcpy(x47, x10, (size_t)(9 * sizeof(float)), hipMemcpyDeviceToHost));
    int x48 = 0;
    while (x48 != 9) {
      printf("%f ", x47[x48]);
      x48 = x48 + 1;
    }
    printf("\n");
    // end copying GPU array x70 to CPU and print for size 9 and type Float
  }
  printf("compile");
  CUDNNCHECK(hipdnnDestroy(x7));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
