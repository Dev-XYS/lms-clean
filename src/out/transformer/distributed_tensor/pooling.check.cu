#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for SGD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    float x35 = x30[x34] * 0.5 + x29[x34];
    x28[x34] = x28[x34] - x35 * 1.0E-4;
    x30[x34] = x35;
    x33 = x33 + x32;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x7;
  CUDNNCHECK(hipdnnCreate(&x7));
  // end setting up the CUDNN environment
  // begin initializing random GPU array of size 162 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(162 * sizeof(float));
  int x9 = 0;
  while (x9 != 162) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(162 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(162 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 162 and type Float at device (pre-rename) x39
  NCCLCHECK(ncclAllReduce(x10, x10, (size_t)(162 * sizeof(float)), ncclFloat32, ncclSum, x4, x5));
  // begin initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(162 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 162);
  // end initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(162 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 162);
  // end initializing fixed GPU array of size 162 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin creating and setting tensor descriptor of shape List(2, 1, 9, 9)
    hipdnnTensorDescriptor_t x36;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x36));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x36, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 9, 9));
    // end creating and setting tensor descriptor
    // begin creating and setting tensor descriptor of shape List(2, 1, 10, 10)
    hipdnnTensorDescriptor_t x37;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x37));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x37, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 10, 10));
    // end creating and setting tensor descriptor
    // begin creating and setting pooling descriptor
    hipdnnPoolingDescriptor_t x38;
    CUDNNCHECK(hipdnnCreatePoolingDescriptor(&x38));
    CUDNNCHECK(hipdnnSetPooling2dDescriptor(x38, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, 2, 2, 1, 1, 1, 1));
    // end creating and setting pooling descriptor
    // begin allocating gpu array for the output of pooling
    CUDA_CALL(hipSetDevice(x6));
    float* x39 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x39, (size_t)(200 * sizeof(float))));
    // end allocating gpu array for the output of pooling
    // begin pooling forward pass
    float x40 = 1.0;
    float x41 = 0.0;
    CUDNNCHECK(hipdnnPoolingForward(x7, x38, &x40, x36, x10, &x41, x37, x39));
    // end pooling forward pass
    // begin initializing fixed GPU array of size 100 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x6));
    float* x42 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x42, (size_t)(100 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x42, 1, 100);
    // end initializing fixed GPU array of size 100 and type Float and device (pre-rename) x39
    // begin allocating gpu array for the gradient of input of pooling
    CUDA_CALL(hipSetDevice(x6));
    float* x43 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x43, (size_t)(162 * sizeof(float))));
    // end allocating gpu array for the gradient of input of pooling
    // begin pooling backward pass
    float x44 = 1.0;
    float x45 = 0.0;
    CUDNNCHECK(hipdnnPoolingBackward(x7, x38, &x44, x37, x39, x37, x42, x36, x10, &x45, x36, x43));
    // end pooling backward pass
    // begin computing ACCUM on GPU for size 162 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x205
    CUDA_CALL(hipSetDevice(x6));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x43, 162);
    // end computing ACCUM on GPU for size 162 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x205
    // begin computing SGD on GPU for size 162 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    CUDA_CALL(hipSetDevice(x6));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 162);
    // end computing SGD on GPU for size 162 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    x19 = x19 + 1;
  }
  if (x6 == 0) {
    // begin copying GPU array x70 to CPU and print for size 162 and type Float
    float* x46 = (float*)malloc(162 * sizeof(float));
    CUDA_CALL(hipMemcpy(x46, x10, (size_t)(162 * sizeof(float)), hipMemcpyDeviceToHost));
    int x47 = 0;
    while (x47 != 162) {
      printf("%f ", x46[x47]);
      x47 = x47 + 1;
    }
    printf("\n");
    // end copying GPU array x70 to CPU and print for size 162 and type Float
  }
  printf("compile");
  CUDNNCHECK(hipdnnDestroy(x7));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
