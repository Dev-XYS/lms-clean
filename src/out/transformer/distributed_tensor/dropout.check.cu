#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "mpi_header.h"
/************* Functions **************/
__global__ void x12(float* x13, float x14, int x15) {
  int x16 = gridDim.x * blockDim.x;
  int x17 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x17 < x15) {
    x13[x17] = x14;
    x17 = x17 + x16;
  }
}
__global__ void x20(float* x21, float* x22, int x23) {
  // begin generating kernel function for ACCUM of type Float
  int x24 = gridDim.x * blockDim.x;
  int x25 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x25 < x23) {
    int x26 = x25;
    x21[x26] = x21[x26] + x22[x26];
    x25 = x25 + x24;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for SGD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    float x35 = x30[x34] * 0.5 + x29[x34];
    x28[x34] = x28[x34] - x35 * 1.0E-4;
    x30[x34] = x35;
    x33 = x33 + x32;
  }
  // end generating kernel function for SGD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin setting up the CUDNN environment
  hipdnnHandle_t x7;
  CUDNNCHECK(hipdnnCreate(&x7));
  // end setting up the CUDNN environment
  // begin initializing random GPU array of size 18 and type Float at device (pre-rename) x39
  float* x8 = (float*)malloc(18 * sizeof(float));
  int x9 = 0;
  while (x9 != 18) {
    x8[x9] = (float)(rand() - RAND_MAX / 2) / (float)RAND_MAX;
    x9 = x9 + 1;
  }
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(18 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x10, x8, (size_t)(18 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing random GPU array of size 18 and type Float at device (pre-rename) x39
  NCCLCHECK(ncclAllReduce(x10, x10, (size_t)(18 * sizeof(float)), ncclFloat32, ncclSum, x4, x5));
  // begin initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x11 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x11, (size_t)(18 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, 0, 18);
  // end initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(18 * sizeof(float))));
  x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x18, 0, 18);
  // end initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
  int x19 = 0;
  while (x19 != 10) {
    // begin initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
    CUDA_CALL(hipSetDevice(x6));
    float* x36 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x36, (size_t)(18 * sizeof(float))));
    x12<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x36, 1, 18);
    // end initializing fixed GPU array of size 18 and type Float and device (pre-rename) x39
    // begin creating and setting tensor descriptor of shape List(2, 1, 3, 3)
    hipdnnTensorDescriptor_t x37;
    CUDNNCHECK(hipdnnCreateTensorDescriptor(&x37));
    CUDNNCHECK(hipdnnSetTensor4dDescriptor(x37, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 2, 1, 3, 3));
    // end creating and setting tensor descriptor
    // begin allocating gpu array for the gradient of input of dropout
    CUDA_CALL(hipSetDevice(x6));
    float* x38 = (float*)malloc(0 * sizeof(float));
    CUDA_CALL(hipMalloc(&x38, (size_t)(18 * sizeof(float))));
    // end allocating gpu array for the gradient of input of dropout
    // begin finding dropout backward reserve bytes
    size_t x39 = (size_t)0;
    CUDNNCHECK(cudnnDropoutGetReserveSpaceSize(x37, &x39));
    // end finding dropout backward reserve bytes
    // begin finding dropout backward states bytes
    size_t x40 = (size_t)0;
    CUDNNCHECK(hipdnnDropoutGetStatesSize(x7, &x40));
    // end finding dropout backward states bytes
    // begin creating dropout descriptor
    hipdnnDropoutDescriptor_t x41;
    CUDNNCHECK(hipdnnCreateDropoutDescriptor(&x41));
    CUDNNCHECK(hipdnnSetDropoutDescriptor(x41, x7, 0.5, x36, x40, 1));
    // end creating dropout descriptor
    // begin dropout backward pass
    CUDNNCHECK(cudnnDropoutBackward(x7, x41, x37, x36, x37, x38, x36, x39));
    // end dropout backward pass
    // begin computing ACCUM on GPU for size 18 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x171
    CUDA_CALL(hipSetDevice(x6));
    x20<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x11, x38, 18);
    // end computing ACCUM on GPU for size 18 and type Float at device (pre-rename) x39 with base_operand x93 and addition_operand x171
    // begin computing SGD on GPU for size 18 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    CUDA_CALL(hipSetDevice(x6));
    x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x11, x18, 18);
    // end computing SGD on GPU for size 18 and type Float at device (pre-name) x39 with weight x70, grad x93, and momentum x131
    x19 = x19 + 1;
  }
  if (x6 == 0) {
    // begin copying GPU array x70 to CPU and print for size 18 and type Float
    float* x42 = (float*)malloc(18 * sizeof(float));
    CUDA_CALL(hipMemcpy(x42, x10, (size_t)(18 * sizeof(float)), hipMemcpyDeviceToHost));
    int x43 = 0;
    while (x43 != 18) {
      printf("%f ", x42[x43]);
      x43 = x43 + 1;
    }
    printf("\n");
    // end copying GPU array x70 to CPU and print for size 18 and type Float
  }
  printf("compile");
  CUDNNCHECK(hipdnnDestroy(x7));
  MPICHECK(MPI_Finalize());
  NCCLCHECK(ncclCommDestroy(x4));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
